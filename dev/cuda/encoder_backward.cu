#include "hip/hip_runtime.h"
/*
Kernels for the positional encoder forward pass in GPT-2.

Compile example:
nvcc -O3 --use_fast_math -lcublas -lcublasLt encoder_backward.cu -o encoder_backward

version 1 is naive port from CPU code to kernel
parallelizes over B,T,C, uses atomics to add to dwte, dwpe
./encoder_backward 1

version 2 is another naive port
parallelizes over C, loops over B,T; much slower than version 1
./encoder_backward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

// GPT-2 positional encoder forward pass
void encoder_backward_cpu(float* dwte, float* dwpe,
                            float* dout, int* inp,
                            int B, int T, int C) {
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            float* dout_bt = dout + b * T * C + t * C;
            int ix = inp[b * T + t];
            float* dwte_ix = dwte + ix * C;
            float* dwpe_t = dwpe + t * C;
            for (int i = 0; i < C; i++) {
                float d = dout_bt[i];
                dwte_ix[i] += d;
                dwpe_t[i] += d;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// naive implementation with atomics
__global__ void encoder_backward_kernel1(float* dwte, float* dwpe,
                                        const float* dout, const int* inp,
                                        int B, int T, int C) {
    /*
    这是一个在 GPU 上执行的全局 kernel，输入参数包含：    
    - dwte：词嵌入（word token embedding）的梯度输出数组    
    - dwpe：位置嵌入（word position embedding）的梯度输出数组    
    - dout：从后续层传回来的梯度数据    
    - inp：输入索引数组，包含每个位置对应的单词索引    
    - B, T, C：分别表示 batch 数、序列长度（time steps）和嵌入维度
    */
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C;

    if (idx < N) {
        int bt = idx / C;
        int b = bt / T;
        int t = bt % T;
        int c = idx % C;
        //利用线性索引 b * T + t 访问输入索引数组 inp，得到当前batch和timestep对应的单词索引 ix。该索引用来定位 dwte 数组中对应单词的梯度位置。
        int ix = inp[b * T + t];

        const float* dout_btc = dout + b * T * C + t * C + c;
        float* dwte_ix = dwte + ix * C + c;
        float* dwpe_tc = dwpe + t * C + c;
        //原子操作 atomicAdd 将 dout_btc 处的梯度加到对应 dwte_ix 和 dwpe_tc 内。  由于多个线程可能同时对同一位置进行累加，为保证数据安全（防止竞态条件），使用 atomicAdd 进行原子性的加法操作。
        atomicAdd(dwte_ix, *dout_btc);
        atomicAdd(dwpe_tc, *dout_btc);
    }
}

// naive implementation that parallelizes over C and loops over B,T
// but it gets rid of atomics
__global__ void encoder_backward_kernel2(float* dwte, float* dwpe,
                                        const float* dout, const int* inp,
                                        int B, int T, int C) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c >= C) { return; } // guard
    int BT = B * T;
    //每一个线程只负责一个维度问题，所以需要循环遍历 B 和 T
    for (int i = 0; i < BT; i++) {
        int t = i % T;
        int ix = inp[i];
        float dout_btc = dout[i * C + c];
        dwte[ix * C + c] += dout_btc;
        dwpe[t * C + c] += dout_btc;
    }
}

// ----------------------------------------------------------------------------
// kernel launcher

void encoder_backward1(float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C,
                    const int block_size) {
    const int N = B * T * C;
    const int grid_size = ceil_div(N, block_size);
    encoder_backward_kernel1<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

void encoder_backward2(float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C,
                    const int block_size) {
    const int grid_size = ceil_div(C, block_size);
    encoder_backward_kernel2<<<grid_size, block_size>>>(dwte, dwpe, dout, inp, B, T, C);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void encoder_backward(int kernel_num,
                     float* dwte, float* dwpe,
                    const float* dout, const int* inp,
                    int B, int T, int C,
                    const int block_size) {
    switch (kernel_num) {
        case 1:
            encoder_backward1(dwte, dwpe, dout, inp, B, T, C, block_size);
            break;
        case 2:
            encoder_backward2(dwte, dwpe, dout, inp, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;
    int V = 50257;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float* dout = make_random_float(B * T * C);
    int* inp = make_random_int(B * T, V);
    float* dwte = make_zeros_float(V * C);
    float* dwpe = make_zeros_float(T * C);

    // move to GPU
    float* d_dout;
    int* d_inp;
    float* d_dwte;
    float* d_dwpe;
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * sizeof(int)));
    cudaCheck(hipMalloc(&d_dwte, V * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dwpe, T * C * sizeof(float)));
    cudaCheck(hipMemcpy(d_dout, dout, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * sizeof(int), hipMemcpyHostToDevice));

    // read kernel_num from command line
    int kernel_num = 1;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // first check the correctness of the kernel
    encoder_backward_cpu(dwte, dwpe, dout, inp, B, T, C);

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        cudaCheck(hipMemset(d_dwte, 0, V * C * sizeof(float)));
        cudaCheck(hipMemset(d_dwpe, 0, T * C * sizeof(float)));
        printf("Checking block size %d.\n", block_size);
        encoder_backward(kernel_num, d_dwte, d_dwpe, d_dout, d_inp, B, T, C, block_size);
        validate_result(d_dwte, dwte, "dwte", V * C, 1e-5f);
        validate_result(d_dwpe, dwpe, "dwpe", T * C, 1e-5f);
    }
    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 1000;
        float elapsed_time = benchmark_kernel(repeat_times, encoder_backward,
                                              kernel_num, d_dwte, d_dwpe, d_dout, d_inp, B, T, C, block_size);
        printf("block_size %4d | time %.4f ms\n", block_size, elapsed_time);
    }

    // free memory
    free(dout);
    free(inp);
    free(dwte);
    free(dwpe);
    hipFree(d_dout);
    hipFree(d_inp);
    hipFree(d_dwte);
    hipFree(d_dwpe);

    return 0;
}
